#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <cstdlib>
#include <string>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

using namespace std;

int THREADS;
int BLOCKS;
int NUM_VALS;

/* Define Caliper region names */
const char* mainFunction = "main";
const char* data_init = "data_init";
const char* correctness_check = "correctness_check ";
const char* comm = "comm";
const char* comm_large = "comm_large";
const char* comm_small = "comm _small";
const char* comp = "comp";
const char* comp_large = "comp_large";
const char* comp_small = "comp_small";


__device__ void swap(int* a, int* b) {
    int t = *a;
    *a = *b;
    *b = t;
}

__device__ int partition(int* arr, int low, int high) {
    int pivot = arr[high];
    int i = (low - 1);

    for (int j = low; j <= high - 1; j++) {
        if (arr[j] < pivot) {
            i++;
            swap(&arr[i], &arr[j]);
        }
    }

    swap(&arr[i + 1], &arr[high]);
    return (i + 1);
}

__global__ void quicksort(int* arr, int low, int high) {
    if (low < high) {
        int pi = partition(arr, low, high);

        if (pi - low < high - pi) {
            quicksort<<<1, 1>>>(arr, low, pi - 1);
            quicksort<<<1, 1>>>(arr, pi + 1, high);
        } else {
            quicksort<<<1, 1>>>(arr, pi + 1, high);
            quicksort<<<1, 1>>>(arr, low, pi - 1);
        }
    }
}

/* Verification */
// CUDA kernel to check if the array is sorted
__global__ void checkArraySorted(int *array, bool *isSorted, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size - 1)
    {
        isSorted[idx] = (array[idx] <= array[idx + 1]);
    }
}
/* Program main */
int main(int argc, char *argv[]) 
{
    int sortingType;

    sortingType = atoi(argv[1]);
    THREADS = atoi(argv[2]);
    NUM_VALS = atoi(argv[3]);
    BLOCKS = NUM_VALS / THREADS;

    printf("Input sorting type: %d\n", sortingType);
    printf("Number of threads: %d\n", THREADS);
    printf("Number of values: %d\n", NUM_VALS);
    printf("Number of blocks: %d\n", BLOCKS);

    CALI_MARK_BEGIN(mainFunction);

    // Create caliper ConfigManager object
    cali::ConfigManager mgr;
    mgr.start();

    CALI_MARK_BEGIN(data_init);
    /* Data generation */
    int *d_unsortedArray;

    // Allocate memory on the GPU and fill
    hipMalloc((void **)&d_unsortedArray, NUM_VALS * sizeof(int));
    generateData<<<BLOCKS, THREADS>>>(d_unsortedArray, NUM_VALS, sortingType);
    hipDeviceSynchronize();
    CALI_MARK_END(data_init);

    /* Main Alg */
    CALI_MARK_BEGIN(comp);
    CALI_MARK_BEGIN(comp_large);
    CALI_MARK_BEGIN(comp_small);
    // Launch the Quicksort kernel
    quicksortLauncher<<<BLOCKS, THREADS>>>(d_unsortedArray, 0, NUM_VALS - 1);
    hipDeviceSynchronize();
    CALI_MARK_END(comp_small);
    CALI_MARK_END(comp_large);
    CALI_MARK_END(comp);

    CALI_MARK_BEGIN(comm);
    CALI_MARK_BEGIN(comm_large);
    CALI_MARK_BEGIN(comm_small);
    // Copy data back to the host
    int sortedArray[NUM_VALS];
    hipMemcpy(sortedArray, d_unsortedArray, NUM_VALS * sizeof(int), hipMemcpyDeviceToHost);
    CALI_MARK_END(comm_small);
    CALI_MARK_END(comm_large);
    CALI_MARK_END(comm);

    CALI_MARK_BEGIN(correctness_check);
    /* Verify Correctness */
    bool isSorted[NUM_VALS - 1];
    bool *d_isSorted;
    hipMalloc((void **)&d_isSorted, (NUM_VALS - 1) * sizeof(bool));
    checkArraySorted<<<BLOCKS, THREADS>>>(d_unsortedArray, d_isSorted, NUM_VALS);
    hipDeviceSynchronize();

    hipMemcpy(isSorted, d_isSorted, (NUM_VALS - 1) * sizeof(bool), hipMemcpyDeviceToHost);

    // Verify if the array is sorted
    bool sorted = true;
    for (int i = 0; i < NUM_VALS - 1; i++)
    {
        if (!isSorted[i])
        {
            sorted = false;
            break;
        }
    }
    CALI_MARK_END(correctness_check);

    // Free GPU memory
    hipFree(d_unsortedArray);
    hipFree(d_isSorted);

    CALI_MARK_END(mainFunction);

    if (sorted)
    {
        printf("The array is sorted!\n");
    }
    else
    {
        printf("The array is not sorted!\n");
    }

    string inputType;
    switch (sortingType)
    {
    case 0:
    {
        inputType = "Randomized";
        break;
    }
    case 1:
    {
        inputType = "Sorted";
        break;
    }
    case 2:
    {
        inputType = "Reverse Sorted";
        break;
    }
    }

    adiak::init(NULL);
    adiak::launchdate();                                                // launch date of the job
    adiak::libraries();                                                 // Libraries used
    adiak::cmdline();                                                   // Command line used to launch the job
    adiak::clustername();                                               // Name of the cluster
    adiak::value("Algorithm", "Quicksort");                             // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", "CUDA");                           // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", "int");                                    // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", sizeof(int));                        // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    adiak::value("InputSize", NUM_VALS);                                // The number of elements in input dataset (1000)
    adiak::value("InputType", inputType);                               // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    adiak::value("num_threads", THREADS);                               // The number of CUDA or OpenMP threads
    adiak::value("num_blocks", BLOCKS);                                 // The number of CUDA blocks
    adiak::value("group_num", 16);                                      // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", "Handwritten, AI, & Online"); // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").

    // Flush Caliper output
    mgr.stop();
    mgr.flush();
}
